#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>
#include "optixHello.h"
#include "helpers.h"
#include "random.h"

//bool is_vpl = false;
#define MIN_DIST2 0.1f//*vpl_dist_scale_square
#define VPL_SHADOW_OFFSET 10.f


extern "C" {
	__constant__ Params params;
}


//_________BASIC FUNCTIONS (UTILS)_______________________________________________________________________
__device__ __inline__ float Clamp(float val, float low, float high) {
	if (val < low) return low;
	else if (val > high) return high;
	else return val;
}
__device__ __inline__ float SmoothStep(float min, float max, float value) {
	float v = Clamp((value - min) / (max - min), 0.f, 1.f);
	return v * v * (-2.f * v + 3.f);
}
__device__ __inline__ float map(float value, float start1, float stop1, float start2, float stop2) {
	return start2 + (stop2 - start2) * ((value - start1) / (stop1 - start1));
}
//_______________________________________________________________________

//------------------GET AND SET RAY INFO------------------
static __device__ __inline__ RadiancePRD getRadiancePRD()
{
	RadiancePRD prd;
	prd.result.x = int_as_float(optixGetPayload_0());
	prd.result.y = int_as_float(optixGetPayload_1());
	prd.result.z = int_as_float(optixGetPayload_2());
	prd.light_number = int_as_float(optixGetPayload_3());
	prd.depth = optixGetPayload_4();
	return prd;
}

static __device__ __inline__ void setRadiancePRD(const RadiancePRD &prd)
{
	optixSetPayload_0(float_as_int(prd.result.x));
	optixSetPayload_1(float_as_int(prd.result.y));
	optixSetPayload_2(float_as_int(prd.result.z));
	optixSetPayload_3(float_as_int(prd.light_number));
	optixSetPayload_4(prd.depth);
}

static __device__ __inline__ OcclusionPRD getOcclusionPRD()
{
	OcclusionPRD prd;
	prd.attenuation.x = int_as_float(optixGetPayload_0());
	prd.attenuation.y = int_as_float(optixGetPayload_1());
	prd.attenuation.z = int_as_float(optixGetPayload_2());
	return prd;
}

static __device__ __inline__ void setOcclusionPRD(const OcclusionPRD &prd)
{
	optixSetPayload_0(float_as_int(prd.attenuation.x));
	optixSetPayload_1(float_as_int(prd.attenuation.y));
	optixSetPayload_2(float_as_int(prd.attenuation.z));
}

//-------------------------------------------------------

//Compute ray value when ray misses the scene
__device__ void phongShadowed()
{
	// this material is opaque, so it fully attenuates all shadow rays
	OcclusionPRD prd;
	prd.attenuation = make_float3(0.f);
	setOcclusionPRD(prd);
	optixTerminateRay();
}

//Function to mark where the VPL are ins the scene
__device__ __inline__ float3 show_VPL(float3 hit_point) {

	float3 vpl_pos_color = make_float3(0.);
	for (int i = 0; i < params.num_hit_vpl; i++)
	{
		VPL showvpl = params.vpls[i];		
		float3 pos_vpl = showvpl.pos;
		float dist = length(hit_point - pos_vpl);
		if (dist < 0.05)
		{
			vpl_pos_color = make_float3(1.);//showvpl.color;// params.cluster_color[params.VPL_assing_cluster[i]];
			//vpl_pos_color = make_float3(1.);
			break;
		}		
	}
	return vpl_pos_color;
}

//Comoute direct illumination of each light source2
__device__ __inline__
float3 direct_light_contribution(float3 hit_point, float3 p_normal, float3 p_Kd, BasicLight current_light) {

	float3 direct_contribution = make_float3(0.);
	BasicLight light = current_light;
	float Ldist = length(light.pos - hit_point);
	float3 L = normalize(light.pos - hit_point);
	float nDl = dot(p_normal, L);
	// cast shadow ray
	float3 light_attenuation = make_float3(static_cast<float>(nDl > 0.0f));
	if (nDl > 0.0f)
	{
		OcclusionPRD shadow_prd;
		shadow_prd.attenuation = make_float3(1.0f);
		optixTrace(
			params.handle,
			hit_point,
			L,
			0.01f,
			Ldist,
			0.0f,
			OptixVisibilityMask(1),
			OPTIX_RAY_FLAG_NONE,
			RAY_TYPE_OCCLUSION,
			RAY_TYPE_COUNT,
			RAY_TYPE_OCCLUSION,
			float3_as_args(shadow_prd.attenuation)/*,
			reinterpret_cast<uint32_t&>(shadow_prd.is_indirect)*/);
		light_attenuation = shadow_prd.attenuation;
	}
	// If not completely shadowed, light the hit point
	if (fmaxf(light_attenuation) > 0.0f)
	{
		float3 Lc = light.color * light_attenuation;
		direct_contribution += p_Kd * nDl * Lc;
	}
	return direct_contribution;
}

//Compute irradiance of each VPL.
__device__ __inline__
float3 VPL_contribution(float3 hit_point, float3 p_normal, float3 p_Kd, VPL current_VPL) {

	float3 irradiance = make_float3(0.);
	//Compute the incident direction of the light coming from the current VPL.
	float3 L = normalize(current_VPL.pos - hit_point);
	//Compute its angle with the point
	float nDl = dot(p_normal, L);
	//If the angle exits between -90 and 90 degrees the VPL can add its contribution
	if (nDl >= 0.0f)
	{
		float Ldist = length(current_VPL.pos - hit_point);//Distance between point and VPL
		float Ldist2 = Ldist * Ldist;// Square of the distane
		//Apply smooth step to aviod aberrations
		float distScale = SmoothStep(.0f + params.minSS, 20.f + params.maxSS, Ldist2);

		//float distScale = Ldist2;
		if (distScale > 0.f)
		{
			float visible;
			OcclusionPRD VPL_prd;

			VPL_prd.attenuation = make_float3(1.0f);
			//Geometric term
			float3 L2 = normalize(hit_point - current_VPL.pos);
			float nvDl2 = dot(current_VPL.normal, L2);
			float G = fabs(nvDl2 * nDl) / Ldist2;// dividod por Ldist2	
			float3 result = current_VPL.color * G  *distScale;

			if (length(result) > 0.02) {
				optixTrace(
					params.handle,
					hit_point,
					L,
					0.01f * VPL_SHADOW_OFFSET,
					Ldist - 0.01,
					0.0f,
					OptixVisibilityMask(1),
					OPTIX_RAY_FLAG_NONE,
					RAY_TYPE_OCCLUSION,
					RAY_TYPE_COUNT,
					RAY_TYPE_OCCLUSION,
					float3_as_args(VPL_prd.attenuation)/*,
					reinterpret_cast<uint32_t&>(shadow_prd.is_indirect)*/);

				visible = VPL_prd.attenuation.x;
			}
			irradiance += result * visible;
		}
	}
	return irradiance;
}

//Compute the total illumination of one point. 
static
__device__ void phongShade(float3 p_Kd,
	float3 p_normal)
{

	//Extract ray and thread information
	const uint3  idx = optixGetLaunchIndex();
	const uint32_t image_index = params.width*idx.y + idx.x;

	const float3 ray_orig = optixGetWorldRayOrigin();
	const float3 ray_dir = optixGetWorldRayDirection();
	const float  ray_t = optixGetRayTmax();
	RadiancePRD prd = getRadiancePRD();

	//Compute hit point
	float3 hit_point = ray_orig + ray_t * ray_dir;

	//Init different types of illumiantion  
	float3 final_direct = make_float3(0, 0, 0);
	float3 final_indirect = make_float3(0.f);

	//Init VPL show color
	float3 vpl_pos_color = make_float3(0.);

	//Show the VPL hit position
	if (params.s_v)
	{
		vpl_pos_color = show_VPL(hit_point);
	}

	//Compute the direct illuminatios
	if (params.s_d)
	{
		for (int ii = 0; ii < params.number_of_lights; ii++)
		{
			BasicLight current_light = params.lights[ii];
			final_direct += direct_light_contribution(hit_point, p_normal, p_Kd, current_light);
		}
	}

	//Comoute indirect illumination

	int  n_vpls = 0;

	if (params.s_i)
	{
		for (int j = 0; j < params.num_hit_vpl; j++)
		{
				VPL vpl = params.vpls[j];//Select VPL			
				n_vpls++;//Know how vpl influence the scene
				//Compute the incident direction of the light coming from the current VPL.
				float3 L = normalize(vpl.pos - hit_point);
				//Compute its angle with the point
				float nDl = dot(p_normal, L);
				//If the angle exits between -90 and 90 degrees the VPL can add its contribution
				if (nDl >= 0.0f)
				{
					float Ldist = length(vpl.pos - hit_point);//Distance between point and VPL
					float Ldist2 = Ldist * Ldist;// Square of the distane
					//Apply smooth step to aviod aberrations
					float distScale = SmoothStep(.0f + params.minSS, 20.f + params.maxSS, Ldist2);

					//float distScale = Ldist2;
					if (distScale > 0.f)
					{
						float visible;
						OcclusionPRD VPL_prd;

						VPL_prd.attenuation = make_float3(1.0f);
						//Geometric term
						float3 L2 = normalize(hit_point - vpl.pos);
						float nvDl2 = dot(vpl.normal, L2);
						float G = fabs(nvDl2 * nDl) / Ldist2;// dividod por Ldist2	

						//if (length(vpl.color * G  *distScale) > 0.05) {
						optixTrace(
							params.handle,
							hit_point,
							L,
							0.01f * VPL_SHADOW_OFFSET,
							Ldist - 0.01,
							0.0f,
							OptixVisibilityMask(1),
							OPTIX_RAY_FLAG_NONE,
							RAY_TYPE_OCCLUSION,
							RAY_TYPE_COUNT,
							RAY_TYPE_OCCLUSION,
							float3_as_args(VPL_prd.attenuation)/*,
							reinterpret_cast<uint32_t&>(shadow_prd.is_indirect)*/);

						visible = VPL_prd.attenuation.x;
						//}											
						final_indirect += vpl.color * G  * visible*distScale;
						//irradiance = make_float3(1,1,1);
					}
				}
			
		}
		final_indirect /= static_cast<float>(n_vpls);
	}
	prd.result = final_direct + (final_indirect * 5) + vpl_pos_color;

	if (params.s_k) {
		int pos_color = params.assing_cluster_vector[image_index];
		prd.result = params.cluster_color[pos_color] / 2 + prd.result / 2;
	}
	
	setRadiancePRD(prd);
}

static
__device__ void compute_R_matrix(float3 p_Kd,
	float3 p_normal)
{
	const uint3    idx = optixGetLaunchIndex();
	int index = idx.x;
	const float3 ray_orig = optixGetWorldRayOrigin();
	const float3 ray_dir = optixGetWorldRayDirection();
	const float  ray_t = optixGetRayTmax();

	float3 hit_point = ray_orig + ray_t * ray_dir;
	float3 irradiance = make_float3(0.f);


	int  n_vpls = 0;


	for (int j = 0; j < params.num_hit_vpl; j++)
	{
		VPL vpl = params.vpls[j];//Select VPL
		//irradiance = make_float3(1, 1, 1);
		params.R_matrix[index*params.num_hit_vpl + j] = make_float3(0.);		
			n_vpls++;//Know how vpl influence the scene
			//Compute the incident direction of the light coming from the current VPL.
			float3 L = normalize(vpl.pos - hit_point);
			//Compute its angle with the point
			float nDl = dot(p_normal, L);
			//If the angle exits between -90 and 90 degrees the VPL can add its contribution
			if (nDl >= 0.0f)
			{
				float Ldist = length(vpl.pos - hit_point);//Distance between point and VPL
				float Ldist2 = Ldist * Ldist;// Square of the distane
				//Apply smooth step to aviod aberrations
				float distScale = SmoothStep(.0f + params.minSS, 20.f + params.maxSS, Ldist2);

				//float distScale = Ldist2;
				if (distScale > 0.f)
				{
					float visible;
					OcclusionPRD VPL_prd;

					VPL_prd.attenuation = make_float3(1.0f);
					//Geometric term
					float3 L2 = normalize(hit_point - vpl.pos);
					float nvDl2 = dot(vpl.normal, L2);
					float G = fabs(nvDl2 * nDl) / Ldist2;// dividod por Ldist2	

					//if (length(vpl.color * G  *distScale) > 0.05) {
					optixTrace(
						params.handle,
						hit_point,
						L,
						0.01f * VPL_SHADOW_OFFSET,
						Ldist - 0.01,
						0.0f,
						OptixVisibilityMask(1),
						OPTIX_RAY_FLAG_NONE,
						RAY_TYPE_OCCLUSION,
						RAY_TYPE_COUNT,
						RAY_TYPE_OCCLUSION,
						float3_as_args(VPL_prd.attenuation)/*,
						reinterpret_cast<uint32_t&>(shadow_prd.is_indirect)*/);

					visible = VPL_prd.attenuation.x;
					//}											
					params.R_matrix[index*params.num_hit_vpl + j] = vpl.color * G  * visible*distScale;		//vpl.color		
				}				
			}			
	}
}

static
__device__ void compute_R_matrix_alt_metric(float3 p_Kd,
	float3 p_normal) {
	const uint3    idx = optixGetLaunchIndex();
	int index = idx.x;
	const float3 ray_orig = optixGetWorldRayOrigin();
	const float3 ray_dir = optixGetWorldRayDirection();
	const float  ray_t = optixGetRayTmax();

	float3 hit_point = ray_orig + ray_t * ray_dir;
	float3 irradiance = make_float3(0.f);
	int  n_vpls = 0;

	for (int j = 0; j < params.num_hit_vpl; j++)
	{
		VPL vpl = params.vpls[j];//Select VPL
		//irradiance = make_float3(1, 1, 1);
		params.R_matrix[index*params.num_hit_vpl + j] = make_float3(0.);
		n_vpls++;//Know how vpl influence the scene
		//Compute the incident direction of the light coming from the current VPL.
		float3 L = normalize(vpl.pos - hit_point);
		//Compute its angle with the point
		float nDl = dot(p_normal, L);
		//If the angle exits between -90 and 90 degrees the VPL can add its contribution
		
		float Ldist = length(vpl.pos - hit_point);//Distance between point and VPL
		float Ldist2 = Ldist * Ldist;// Square of the distane
		//Apply smooth step to aviod aberrations
		float distScale = SmoothStep(.0f + params.minSS, 20.f + params.maxSS, Ldist2);

		//float distScale = Ldist2;		
		float visible;
		OcclusionPRD VPL_prd;

		VPL_prd.attenuation = make_float3(1.0f);
		//Geometric term
		float3 L2 = normalize(hit_point - vpl.pos);
		float nvDl2 = dot(vpl.normal, L2);
		float G = fabs(nvDl2 * nDl) / Ldist2;// dividod por Ldist2	

		optixTrace(
			params.handle,
			hit_point,
			L,
			0.01f * VPL_SHADOW_OFFSET,
			Ldist - 0.01,
			0.0f,
			OptixVisibilityMask(1),
			OPTIX_RAY_FLAG_NONE,
			RAY_TYPE_OCCLUSION,
			RAY_TYPE_COUNT,
			RAY_TYPE_OCCLUSION,
			float3_as_args(VPL_prd.attenuation)
			);

		visible = VPL_prd.attenuation.x;
															
		params.R_matrix[index*params.num_hit_vpl + j] = vpl.color * p_Kd * dot(p_normal, L) * dot(vpl.normal, -L) * visible;
	}
		
}
static
__device__ void result_K_means(float3 p_Kd,
	float3 p_normal) {

	const uint3    idx = optixGetLaunchIndex();
	const float3 ray_orig = optixGetWorldRayOrigin();
	const float3 ray_dir = optixGetWorldRayDirection();
	const float  ray_t = optixGetRayTmax();

	const uint32_t image_index = params.width*idx.y + idx.x;


	RadiancePRD prd = getRadiancePRD();

	float3 hit_point = ray_orig + ray_t * ray_dir;
	bool vplhit = false;

	float3 vpl_pos_col = make_float3(0.);
	float3 final_direct = make_float3(0, 0, 0);
	float3 irradiance = make_float3(0.f);

	
	//Show the VPL hit position
	if (params.s_v)
	{
		if (params.show_cluster_VPL) {
			for (int j = 0; j < MAX_VPL_CLUSTERS; j++)/* for (int j = 0; j < params.num_vpl*(params.max_bounces + 1); j++)*/
			{
				int point_cluster = params.assing_cluster_vector[image_index];
				int vpl_pos = params.selected_VPL_pos[point_cluster * MAX_VPL_CLUSTERS + j];
				VPL showvpl = params.vpls[vpl_pos];
				if (showvpl.hit)
				{
					float3 pos_vpl = showvpl.pos;
					float dist = length(hit_point - pos_vpl);
					if (dist < 0.05)
					{
						//vpl_pos = showvpl.color / 4;
						//vpl_pos = make_float3(1,1,1);
						vpl_pos_col = showvpl.color;// make_float3(1.);
						//vpl_pos = params.cluster_color[0];
						vplhit = true;
						break;
					}
				}
			}
		}
		if (params.show_cluster_VPL == false) {
			for (int i = 0; i < params.num_hit_vpl; i++)
			{
				VPL showvpl = params.vpls[i];
				float3 pos_vpl = showvpl.pos;
				float dist = length(hit_point - pos_vpl);
				if (dist < 0.05)
				{
					vpl_pos_col = params.cluster_color[params.VPL_assing_cluster[i]];
					//vpl_pos_color = make_float3(1.);
					break;
				}
			}
		}
		
	}

	
		//Compute the direct illuminatios
		if (params.s_d)
		{
			for (int ii = 0; ii < params.number_of_lights; ii++)
			{
				BasicLight current_light = params.lights[ii];
				final_direct += direct_light_contribution(hit_point, p_normal, p_Kd, current_light);
			}
		}	

		//Indirect VPL irradiance
		int  n_vpls = 1.f;
		if (params.s_i)
		{
			for (int j = 0; j < MAX_VPL_CLUSTERS; j++)/* for (int j = 0; j < params.num_vpl*(params.max_bounces + 1); j++)*/
			{
				int point_cluster = params.assing_cluster_vector[image_index];
				int vpl_pos = params.selected_VPL_pos[point_cluster * MAX_VPL_CLUSTERS + j];
				VPL vpl = params.vpls[vpl_pos];//Select VPL   -- 
				//vpl = params.vpls[j];

				//irradiance = make_float3(1, 1, 1);
				//if (params.vpls[params.first_VPL_cluster[params.VPL_assing_cluster[vpl_pos]]].hit == true) {
				if (vpl.hit)
				{
					n_vpls++;//Know how vpl influence the scene
					//Compute the incident direction of the light coming from the current VPL.
					float3 L = normalize(vpl.pos - hit_point);
					//Compute its angle with the point
					float nDl = dot(p_normal, L);
					//If the angle exits between -90 and 90 degrees the VPL can add its contribution
					if (nDl >= 0.0f)
					{
						float Ldist = length(vpl.pos - hit_point);//Distance between point and VPL
						float Ldist2 = Ldist * Ldist;// Square of the distane
						//Apply smooth step to aviod aberrations
						float distScale = SmoothStep(.0f + params.minSS, 20.f + params.maxSS, Ldist2);

						//float distScale = Ldist2;
						if (distScale > 0.f)
						{
							float visible;
							OcclusionPRD VPL_prd;

							VPL_prd.attenuation = make_float3(1.0f);
							//Geometric term
							float3 L2 = normalize(hit_point - vpl.pos);
							float nvDl2 = dot(vpl.normal, L2);
							float G = fabs(nvDl2 * nDl) / Ldist2;// dividod por Ldist2	

							//if (length(vpl.color * G  *distScale) > 0.05) {
							optixTrace(
								params.handle,
								hit_point,
								L,
								0.01f * VPL_SHADOW_OFFSET,
								Ldist - 0.01,
								0.0f,
								OptixVisibilityMask(1),
								OPTIX_RAY_FLAG_NONE,
								RAY_TYPE_OCCLUSION,
								RAY_TYPE_COUNT,
								RAY_TYPE_OCCLUSION,
								float3_as_args(VPL_prd.attenuation)/*,
								reinterpret_cast<uint32_t&>(shadow_prd.is_indirect)*/);

							visible = VPL_prd.attenuation.x;
							//}											
							irradiance += vpl.color * G  * visible*distScale;
							//irradiance = make_float3(1,1,1);
						}
					}
				}
				//}

			}
			irradiance /= static_cast<float>(n_vpls);
			//irradiance = p_Kd * irradiance;
		}
		//Direct;		
		// pass the color back
		prd.result = final_direct + (irradiance * 10) + vpl_pos_col;

		if (params.s_k) {
			int pos_color = params.assing_cluster_vector[image_index];
			prd.result = params.cluster_color[pos_color] / 8 + prd.result / 2;
		}


		setRadiancePRD(prd);
	


}

extern "C" __global__ void __closesthit__diffuse_radiance()
{
	

	float3 object_normal = make_float3(
		int_as_float(optixGetAttribute_0()),
		int_as_float(optixGetAttribute_1()),
		int_as_float(optixGetAttribute_2()));

	float3 world_normal = normalize(optixTransformNormalFromObjectToWorldSpace(object_normal));
	float3 ffnormal = faceforward(world_normal, -optixGetWorldRayDirection(), world_normal);	   

	const HitGroupData* sbt_data = (HitGroupData*)optixGetSbtDataPointer();
	const Phong &phong = sbt_data->shading.diffuse;

	if (params.compute_image && !params.result_K_means) {
		phongShade(phong.Kd, ffnormal);	
	}

	if (params.select_space_points) {

		const uint3    idx = optixGetLaunchIndex();
		const float3 ray_orig = optixGetWorldRayOrigin();
		const float3 ray_dir = optixGetWorldRayDirection();
		const float  ray_t = optixGetRayTmax();

		const uint32_t image_index = params.width*idx.y + idx.x;

		float3 hit_point = ray_orig + ray_t * ray_dir;
		params.normal[image_index] = make_float3(ffnormal.x, ffnormal.y, ffnormal.z);
		params.pos[image_index] = make_float3(hit_point.x, hit_point.y, hit_point.z);
	}
	if (params.compute_R) {
		//compute_R_matrix(phong.Kd, ffnormal);
		compute_R_matrix_alt_metric(phong.Kd, ffnormal);
	}
	if (params.compute_image && params.result_K_means) {
		result_K_means(phong.Kd,ffnormal);

	}
	
}


extern "C" __global__ void __anyhit__full_occlusion()
{
		phongShadowed();	
}





extern "C" __global__ void __miss__constant_bg()
{	
	if (params.compute_image) {

		const MissData* sbt_data = (MissData*)optixGetSbtDataPointer();
		RadiancePRD prd = getRadiancePRD();
		prd.result = sbt_data->bg_color;
		setRadiancePRD(prd);

		}
	if (params.select_space_points) {
		const uint3 idx = optixGetLaunchIndex();
		const uint32_t image_index = params.width*idx.y + idx.x;

		params.normal[image_index] = make_float3(0, 0, 0);
		params.pos[image_index] = make_float3(1000, 1000, 1000);
	}
		

}

